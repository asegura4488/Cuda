#include <hip/hip_runtime.h>
#include <stdio.h>
#include <memory>
#include <iostream> 

const int Nthreads = 1024;
const int Nentries = 2048;
const int rows = 30;
const int columns = 30;
//const int blocksize = 16; 
 
__global__ 
void add_vectors(float *A, float *B, float *C, int N, int M, int L ) 
{
    int tx = threadIdx.x + blockDim.x*blockIdx.x;
	int ty = threadIdx.y + blockDim.y*blockIdx.y;
    float valor = 0.f;
    for (int i = 0; i<L; ++i){
    valor += A[ty*L+i]*B[i*M+tx];
    }
    C[ty*M+tx]=valor;			
}
 
int main()
{
        int N = rows;
        int M = columns;
        int L = 2;
        int *Nd, *Md, *Ld;
        float A[rows][columns], B[rows][columns], C[rows][columns];
        float *dev_a, *dev_b, *dev_c;

	hipMalloc((void **) &dev_a, rows*columns*sizeof(int));
    hipMalloc((void **) &dev_b, rows*columns*sizeof(int));
	hipMalloc((void **) &dev_c, rows*columns*sizeof(int));
	hipMalloc((void **) &Nd, N*sizeof(int));
	hipMalloc((void **) &Md, M*sizeof(int));
	hipMalloc((void **) &Ld, L*sizeof(int));

    hipMemcpy( Nd, &N, N*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( Md, &M, M*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( Md, &L, M*sizeof(int), hipMemcpyHostToDevice );
          
	// Fill Arrays
        for (int y = 0; y < rows; y++)
	for (int x = 0; x < columns; x++)
	{
	A[y][x] = 1.+float(x)+float(y);
	B[y][x] = 1.+float(y)+float(x+1);
	}
	

	hipMemcpy( dev_a, &A, rows*columns*sizeof(int), hipMemcpyHostToDevice ); 
	hipMemcpy( dev_b, &B, rows*columns*sizeof(int), hipMemcpyHostToDevice ); 


    dim3 dimGrid( Nentries/Nthreads+(Nentries % Nthreads ? 1 : 0), 2 );
    dim3 dimBlock( rows, columns );

 	add_vectors<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N, M, L);
 	

        hipMemcpy( &C, dev_c, rows*columns*sizeof(int), hipMemcpyDeviceToHost ); 
	
 	hipFree(dev_a);
        hipFree(dev_b);
 	hipFree(dev_c);
       
        for (int y = 0; y < rows; y++)
	for (int x = 0; x < columns; x++)
	{
	std::cout << "A["<<y<<"]["<<x<<"]= " << A[x][y] << " B["<<y<<"]["<<x<<"]= "  <<
        B[y][x] << " C["<<y<<"]["<<x<<"]= " << C[y][x]<< std::endl;
	}
        
	return EXIT_SUCCESS;
	//return 0;

	
}
