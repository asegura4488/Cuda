
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream> 

const int Nthreads = 1024;
const int Nentries = 2500;
 
__global__ 
void add_vectors(int *a, int *b, int *c) 
{

   // int tID = threadIdx.x;
    int tID = threadIdx.x + blockDim.x*blockIdx.x;  
	if (tID < Nentries){
	c[tID] = a[tID] + b[tID];
	}
}
 
int main()
{
        int a[Nentries], b[Nentries], c[Nentries]={0};
        int *dev_a, *dev_b, *dev_c;

	hipMalloc((void **) &dev_a, Nentries*sizeof(int));
    hipMemset(dev_a, 0, Nentries*sizeof(int));
    hipMalloc((void **) &dev_b, Nentries*sizeof(int));
    hipMemset(dev_b, 0, Nentries*sizeof(int));
	hipMalloc((void **) &dev_c, Nentries*sizeof(int));
    hipMemset(dev_c, 0, Nentries*sizeof(int));
	
	// Fill Arrays
	for (int i = 0; i < Nentries; i++)
	{
	a[i] = i;
	b[i] = i-1;
	}

	hipMemcpy( dev_a, &a, Nentries*sizeof(int), hipMemcpyHostToDevice ); 
	hipMemcpy( dev_b, &b, Nentries*sizeof(int), hipMemcpyHostToDevice ); 
	
    dim3 dimGrid( Nentries/Nthreads+(Nentries % Nthreads ? 1 : 0), 1 );
    dim3 dimBlock( Nthreads, 1 );

    add_vectors<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize(); 
    hipMemcpy( &c, dev_c, Nentries*sizeof(int), hipMemcpyDeviceToHost ); 
	
 	hipFree(dev_a);
    hipFree(dev_b);
 	hipFree(dev_c);
       
        for (int i = 0; i < Nentries; i++)
	{
	std::cout << "a[i]=" << a[i] << " b[i]=" << b[i] << " c[i]=" << c[i] << std::endl; 
	}
	return EXIT_SUCCESS;
	//return 0;

	
}
