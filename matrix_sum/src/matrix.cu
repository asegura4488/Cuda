
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream> 

const int rows = 10;
const int columns = 20;
//const int blocksize = 16; 
 
__global__ 
void add_vectors(int *a, int *b, int *c) 
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int i = (columns*y) + x;
	c[i] = a[i] + b[i];
			
}
 
int main()
{
        int a[rows][columns], b[rows][columns], c[rows][columns];
        int *dev_a, *dev_b, *dev_c;

	hipMalloc((void **) &dev_a, rows*columns*sizeof(int));
        hipMalloc((void **) &dev_b, rows*columns*sizeof(int));
	hipMalloc((void **) &dev_c, rows*columns*sizeof(int));
	
          
	// Fill Arrays
        for (int y = 0; y < rows; y++)
	for (int x = 0; x < columns; x++)
	{
	a[y][x] = x;
	b[y][x] = y;
	}
	

	hipMemcpy( dev_a, &a, rows*columns*sizeof(int), hipMemcpyHostToDevice ); 
	hipMemcpy( dev_b, &b, rows*columns*sizeof(int), hipMemcpyHostToDevice ); 
	
        dim3 grid(columns,rows);
        //dim3 dimBlock( N, 1 );
        //dim3 dimGrid( 1, 1 );
 	add_vectors<<<grid, 1>>>(dev_a, dev_b, dev_c);
 	

        hipMemcpy( &c, dev_c, rows*columns*sizeof(int), hipMemcpyDeviceToHost ); 
	
 	hipFree(dev_a);
        hipFree(dev_b);
 	hipFree(dev_c);
       
        for (int y = 0; y < rows; y++)
	for (int x = 0; x < columns; x++)
	{
	std::cout << "y " << y << " x " << x << std::endl;
	std::cout << "a[y][x] " << a[y][x]  << " b[y][x] " << b[y][x] << " c[y][x] " << c[y][x]<< std::endl;
	}
        
	return EXIT_SUCCESS;
	//return 0;

	
}
